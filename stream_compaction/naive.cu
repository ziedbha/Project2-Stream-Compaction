#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
	namespace Naive {
		int* dev_bufIn;
		int* dev_bufOut;

		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__global__ void kernNaiveScan(int n, int level, int* g_odata, int* g_idata) {
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index >= n) {
				return;
			}
			int offset = (int)powf(2.0f, (float)(level - 1));
			if (index >= offset) {
				g_odata[index] = g_idata[index - offset] + g_idata[index];
			}
			else {
				g_odata[index] = g_idata[index];
			}
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {
			// malloc device buffers
			hipMalloc((void**)&dev_bufIn, n * sizeof(int));
			checkCUDAError("CUDA Malloc error!");
			hipMalloc((void**)&dev_bufOut, n * sizeof(int));
			checkCUDAError("CUDA Malloc error!");

			// copy input data into device
			hipMemcpy(dev_bufIn, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("CUDA Memcpy error!");


			dim3 numberOfBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
			timer().startGpuTimer();
			// placeholder pointers to do dual buffering
			int* input = dev_bufIn;
			int* output = dev_bufOut;
			for (int i = 1; i <= ilog2ceil(n); i++) {
				kernNaiveScan << <numberOfBlocks, BLOCK_SIZE >> > (n, i, output, input);

				// swap buffers
				int* temp = output;
				output = input;
				input = temp;

				hipDeviceSynchronize();
			}


			// the above scan is inclusive --> run the conversion kernel
			Common::kernInclusiveToExclusive << <numberOfBlocks, BLOCK_SIZE >> > (n, output, input);
			timer().endGpuTimer();
			if (output == dev_bufOut) {
				hipMemcpy(odata, dev_bufOut, n * sizeof(int), hipMemcpyDeviceToHost);
				checkCUDAError("CUDA Memcpy error!");
			}
			else {
				hipMemcpy(odata, dev_bufIn, n * sizeof(int), hipMemcpyDeviceToHost);
				checkCUDAError("CUDA Memcpy error!");
			}

			// free malloc'd device memory
			hipFree(dev_bufIn);
			hipFree(dev_bufOut);
		}
	}
}
